#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  cuda_support.cu
 *
 *    Description:  This file contains the cuda functions which provides  general
 *                    support of the task and have nothing to do with the
 *                    specific algorithm and calculation
 *                    Mostly the interface for other cpp source file
 *
 *        Created:  Thu Jul 23 03:38:40 2015
 *       Modified:  Sat Sep  5 10:55:08 2015
 *
 *         Author:  Huang Zonghao
 *          Email:  coding@huangzonghao.com
 *
 * =============================================================================
 */

#include "../include/cuda_support.h"

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../thirdparty/nvidia/hip/hip_runtime_api.h"
#include "../thirdparty/nvidia/hip/hip_vector_types.h"

#include "../include/cuda_support-inl.h"
#include "../include/demand_distribution.h"


/* =============================================================================
 *  The device kernels
 * =========================================================================== */



/* =============================================================================
 *  The global kernels
 * =========================================================================== */


/*
 * ===  GLOBAL KERNEL  =========================================================
 *         Name:  g_ZeroizeMemoryFloat
 *  Description:  zeroize the float array
 *       @param:  pointer to the array, length
 * =============================================================================
 */
__global__
void g_ZeroizeMemoryFloat(float *array, size_t length){
    size_t step_size = gridDim.x * blockDim.x;
    size_t myStartIdx = blockDim.x * blockIdx.x + threadIdx.x;
    for (size_t i = myStartIdx; i < arrayLength; i += step_size)
        array[i] = 0;

    __syncthreads();
    return;
}       /* -----  end of global kernel g_ZeroizeMemoryFloat  ----- */

/* =============================================================================
 *  The host functions
 * =========================================================================== */
/*
 * ===  FUNCTION  ==============================================================
 *         Name:  cuda_PassToDevice
 *  Description:  pass the array to device
 *       @param:  pointer to host array, pointer to device array, array size
 *      @return:  void
 * =============================================================================
 */
void cuda_PassToDevice ( const float *h_array,
                         const float *d_array,
                         size_t length ){

    checkCudaErrors(hipMemcpy(d_array, h_array,
                               length * sizeof(float),
                               hipMemcpyHostToDevice));
    return;
}       /* -----  end of function cuda_PassToDevice  ----- */

/* reload */
void cuda_PassToDevice ( const float **h_array,
                         const float **d_array,
                         size_t length ){

    checkCudaErrors(hipMemcpy(d_array, h_array,
                               length * sizeof(float *),
                               hipMemcpyHostToDevice));
    return;
}       /* -----  end of function cuda_PassToDevice  ----- */

/* reload */
void cuda_PassToDevice ( const size_t *h_array,
                         const size_t *d_array,
                         size_t length ){

    checkCudaErrors(hipMemcpy(d_array, h_array,
                               length * sizeof(size_t),
                               hipMemcpyHostToDevice));
    return;
}       /* -----  end of function cuda_PassToDevice  ----- */

/* reload */
void cuda_PassToDevice ( const struct DemandDistribution *h_array,
                         const struct DemandDistribution *d_array,
                         size_t length ){

    checkCudaErrors(hipMemcpy(d_array, h_array,
                               length * sizeof(struct DemandDistribution),
                               hipMemcpyHostToDevice));
    return;
}       /* -----  end of function cuda_PassToDevice  ----- */

void cuda_PassToDevice ( const struct DemandDistribution **h_array,
                         const struct DemandDistribution **d_array,
                         size_t length ){

    checkCudaErrors(hipMemcpy(d_array, h_array,
                               length * sizeof(struct DemandDistribution*),
                               hipMemcpyHostToDevice));
    return;
}       /* -----  end of function cuda_PassToDevice  ----- */

/*
 * ===  FUNCTION  ==============================================================
 *         Name:  cuda_ReadFromDevice
 *  Description:  read the array from device
 *       @param:  see pass_to_array
 *      @return:  see pass_to_array
 * =============================================================================
 */
void cuda_ReadFromDevice ( const float *h_array,
                           const float *d_array,
                           size_t length ){

    checkCudaErrors(hipMemcpy(h_array, d_array,
                               length * sizeof(float),
                               hipMemcpyDeviceToHost));

    return ;
}       /* -----  end of function cuda_ReadFromDevice  ----- */

/* reload */
void cuda_ReadFromDevice ( const size_t *h_array,
                           const size_t * d_array,
                           size_t length ){

    checkCudaErrors(hipMemcpy(h_array, d_array,
                               length * sizeof(size_t),
                               hipMemcpyDeviceToHost));

    return ;
}       /* -----  end of function cuda_ReadFromDevice  ----- */

/*
 * ===  FUNCTION  ==============================================================
 *         Name:  cuda_AllocateMemory
 *  Description:  allocate the memory to the given pointer (the data type is
 *                  float since we only need to take care of the value table
 *                  and the distribution table)
 *       @param:  length
 *      @return:  float*
 * =============================================================================
 */
float *cuda_AllocateMemoryFloat(size_t length){
    float *temp;
    checkCudaErrors(hipMalloc(&temp, length * sizeof(float)));
    return temp;
}       /* -----  end of function cuda_AllocateMemoryFLoat  ----- */


int *cuda_AllocateMemoryInt(size_t length){
    int *temp;
    checkCudaErrors(hipMalloc(&temp, length * sizeof(int)));
    return temp;
}

float **cuda_AllocateMemoryFloatPtr(size_t length){
    float **temp;
    checkCudaErrors(hipMalloc(&temp, length * sizeof(float*)));
    return temp;
}

struct DemandDistribution *cuda_AllocateMemoryDemandDistribution(size_t length){
    struct DemandDistribution *temp;
    checkCudaErrors(hipMalloc(&temp, length * sizeof(struct DemandDistribution)));
    return temp;
}

struct DemandDistribution **cuda_AllocateMemoryDemandDistributionPtr(size_t length){
    struct DemandDistribution **temp;
    checkCudaErrors(hipMalloc(&temp, length * sizeof(struct DemandDistribution*)));
    return temp;
}

/*
 * ===  FUNCTION  ==============================================================
 *         Name:  cuda_ZeroizeMemoryFloat
 *  Description:  zeroize the float array
 *       @param:  pointer to array, array length
 *      @return:  success or not
 * =============================================================================
 */
bool cuda_ZeroizeMemoryFloat(float *array, size_t length){
    <+body+>
        return <+return value+>;
}       /* -----  end of function cuda_ZeroizeMemoryFloat  ----- */

/*
 * ===  FUNCTION  ==============================================================
 *         Name:  cuda_FreeMemory
 *  Description:  free the cuda memory holding by the given pointer
 *       @param:  pointer
 *      @return:  none
 * =============================================================================
 */
void cuda_FreeMemory(float *ptr){
    checkCudaErrors(hipFree(ptr));
    return;
}

void cuda_FreeMemory(float **ptr){
    checkCudaErrors(hipFree(ptr));
    return;
}

void cuda_FreeMemory(struct DemandDistribution *ptr){
    checkCudaErrors(hipFree(ptr));
    return;
}

void cuda_FreeMemory(struct DemandDistribution **ptr){
    checkCudaErrors(hipFree(ptr));
    return;
}
/* -----  end of function cuda_FreeMemory  ----- */

/*
 * ===  FUNCTION  ==============================================================
 *         Name:  cuda_CheckGPU
 *  Description:  returns the number of devices, number of blocks per device,
 *                  and number of threads per block
 *       @param:  the pointer to the three parameters
 *      @return:  none
 * =============================================================================
 */
bool cuda_CheckGPU(int *num_devices, int *num_cores, int *core_size){
    hipGetDeviceCount(*num_devices);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    *num_cores = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *\
                                      deviceProp.multiProcessorCount;
    *core_size = deviceProp.maxThreadsPerBlock;
    return true;
}       /* -----  end of function cuda_CheckGPU  ----- */

/* =============================================================================
 *                         end of file cuda_support.cu
 * =============================================================================
 */
