#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  model_dp.cu
 *
 *    Description:  All the functions to compute with the dynamic programming
 *                    algorithm
 *
 *        Created:  Fri Aug  7 23:47:24 2015
 *       Modified:  Wed Aug 26 17:23:40 2015
 *
 *         Author:  Huang Zonghao
 *          Email:  coding@huangzonghao.com
 *
 * =============================================================================
 */
#include "../include/models.h"
#include "../include/model_support.h"
#include "../include/demand_distribution.h"

/*
 * ===  GLOBAL KERNEL  =========================================================
 *         Name:  g_ModelDPInit
 *  Description:  init the DP table with the tree structrue
 *       @param:  current table, the index of the current level index, the total
 *                   number of states contained in this level and the batch index
 *                   for the current level (how many turns that the kernel has
 *                   been working on this level)
 * =============================================================================
 */

/* :REMARKS:Tue Aug 25 19:28:15 2015:huangzonghao:
 *  for each level, we are gonna calculate from the 1 to k for some digit
 */
__global__
void g_ModelDPInit(float *current_table,
                   size_t batchIdx,
                   size_t level_size,
                   float s ){
    /* myIdx is the index of the current state within each level */
    size_t myIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if ( myIdx < level_size ){
        size_t current_data_idx = batchIdx * level_size + myIdx;
        size_t parent_data_idx = current_data_idx - level_size;
        if(current_data_idx == 0){
            current_table[current_data_idx] = 0.0;
        }
        else {
            current_table[current_data_idx] = current_table[parent_data_idx] + s;
        }
    }
    return;
}       /* -----  end of global kernel g_ModelDPInit  ----- */

/*
 * ===  FUNCTION  ==============================================================
 *         Name:  ModelDPInit
 *  Description:  The initialization function for ModelDP
                      to calculate all the state value from the boundary conditions
 *       @param:  the control sequence, the system information
 *      @return:  success or not
 * =============================================================================
 */
bool ModelDPInit(CommandQueue * cmd, SystemInfo * sysinfo, float *value_table){
    /* the first layer with level zero and level size one */
    g_ModelDPInit<<<1,1>>>(value_table, 0, 1);
    /* then the each level just get larger and larger */
    size_t level_size;
    size_t num_blocks_used;
    size_t core_size = sysinfo->get_value("core_size");
    for(int i_level = 0; i_level < cmd->get_h_params("m"); ++i_level){
        level_size = pow(cmd->get_h_params("m"), i_level);
        num_blocks_used = level_size / core_size + 1;
        for(int i_batch = 1; i_batch < cmd->get_h_params("k"); ++i_batch){
            g_ModelDPInit<<<num_blocks_used, core_size>>>(value_table,
                                                          i_batch,
                                                          level_size,
                                                          cmd->get_h_params("s"));
        }
    }
    return true;
}       /* -----  end of function ModelDPInit  ----- */

/*
 * ===  GLOBAL KERNEL  =========================================================
 *         Name:  g_ModelDP
 *  Description:  the kernel function for the tree structure
 *       @param:  table to update, table for reference, amount to deplete, amount
 *                   to order, the total number of states of this level the
 *                   batch index(the value of the digit we are updating within
 *                   certain level)
 * =============================================================================
 */
__global__
void g_ModelDP(float *table_to_update,
               float *table_for_reference,
               int *z_records,
               int *q_records,
               size_t level_size,
               size_t batchIdx,
               DeviceParameters d){

    size_t myIdx = threadIdx.x + blockIdx.x * blockDim.x;

    if (myIdx < level_size) {

        size_t dataIdx = batch_idx * level_size + myIdx;
        size_t parentIdx = dataIdx - level_size;

        if (current == 0 || depletion[parent] == 0) {
            d_StateValueUpdate(table_to_update,
                               table_for_reference,
                               dataIdx,
                               z_records, q_records,
                               /* [min_z, max_z] */
                               0, 2,
                               /* [min_q, max_q] */
                               0, k - 1,
                               0, d);
        }
        else /* (depletion[parent] != 0) */ {
            d_StateValueUpdate(table_to_update,
                               table_for_reference,
                               z_records, q_records,
                               dataIdx,
                               /* [min_z, max_z] */
                               z_records[parentIdx] + 1,
                               z_records[parentIdx] + 2,
                               /* [min_q, max_q] */
                               q_records[parentIdx],
                               q_records[parentIdx] + 1,
                               0, d);
        }
    }
    return;
}       /* -----  end of global kernel g_ModelDP  ----- */
/*
 * ===  FUNCTION  ==============================================================
 *         Name:  ModelDP
 *  Description:  to update the table for one period with the dynamic programming
 *                  algorithm
 *       @param:  control sequence, system information, the index of the period
 *      @return:  success or not
 * =============================================================================
 */
bool ModelDP(CommandQueue *cmd,
             SystemInfo *sysinfo,
             float *table_to_update,
             float *table_for_reference,
             int *z, int *q){

    size_t level_size = pow(cmd->get_h_params("k"), cmd->get_h_params("m"));
    // The very first state 0,0,...,0
    g_ModelDP<<<1, 1>>>(  table_to_update,
                          table_for_reference,
                          z, q,
                          1, 0
                          *(cmd->get_device_param_pointer) );

    size_t num_blocks_used;
    size_t core_size = sysinfo->get_value("core_size");
    for (size_t i_level = 0; i_level < cmd->get_h_params("m"); ++i_level) {
        num_blocks_used = i_level * cmd->get_h_params("k");
        for (size_t i_batch = 1; i_batch < n_capacity; i_batch++) {
            g_ModelDP<<<num_blocks_used, core_size >>>(  table_to_update,
                                                         table_for_reference,
                                                         z, q,
                                                         level_size,
                                                         i_batch,
                                                         *(cmd->get_device_param_pointer) );
        }
    }
    return true;
}       /* -----  end of function ModelDP  ----- */


/* =============================================================================
 *                         end of file model_dp.cu
 * =============================================================================
 */
