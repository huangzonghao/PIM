#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  model_dp.cu
 *
 *    Description:  All the functions to compute with the dynamic programming
 *                    algorithm
 *
 *        Created:  Fri Aug  7 23:47:24 2015
 *       Modified:  Sat Aug  8 12:16:58 2015
 *
 *         Author:  Huang Zonghao
 *          Email:  coding@huangzonghao.com
 *
 * =============================================================================
 */
#include "../include/models.h"

 // Helper function to get CUDA thread id
// whenever we use __device__ function
__device__ inline size_t
get_thread_id() {

    size_t blockId = blockIdx.x +
                     blockIdx.y * gridDim.x +
                     gridDim.x * gridDim.y * blockIdx.z;
    return blockId * blockDim.x + threadIdx.x;
}


// Using these values for general CUDA GPU is just fine
inline void
get_grid_dim(dim3* block_dim, dim3* grid_dim, size_t batch_size) {

    size_t n_block = batch_size / 512 + 1;

    assert(block_dim && grid_dim);
    *block_dim = dim3(512, 1, 1);
    *grid_dim = dim3(4096, n_block / 4096 + 1, 1);
}


// CUDA Kernel function for initialization
__global__ void
init_kernel(float *current_values,
            size_t batch_idx,
            size_t batch_size) {

    size_t thread_idx = get_thread_id();

    if (thread_idx < batch_size) {

        size_t current = batch_idx * batch_size + thread_idx;
        size_t parent = current - batch_size;

        if (current == 0) {
            current_values[current] = 0.0;
        } else {
            current_values[current] = current_values[parent] + 1.0;
        }
    }
}


// Plain C function for interact with kernel
void
init_states(float *current_values) {

    size_t num_states = std::pow(n_capacity, n_dimension);

    // The very first state
    init_kernel<<<1, 1>>>(current_values, 0, 1);

    for (size_t d = 0; d < n_dimension; d++) {

        size_t batch_size = pow(n_capacity, d);

        dim3 block_dim, grid_dim;
        get_grid_dim(&block_dim, &grid_dim, batch_size);

        for (size_t batch_idx = 1; batch_idx < n_capacity; batch_idx++) {
            init_kernel<<<grid_dim, block_dim>>>(current_values,
                                                 batch_idx,
                                                 batch_size);
        }
    }

    hipDeviceSynchronize();
    hipDeviceSynchronize();

}

// The CUDA kernel function for DP_news
__global__ void
iter_kernel(float *current_values,
            dp_int *depletion,
            dp_int *order,
            float *future_values,
            int period,
            size_t batch_idx,
            size_t batch_size) {

    size_t thread_idx = get_thread_id();

    if (thread_idx < batch_size) {
       // first update current_values

        size_t current = batch_idx * batch_size + thread_idx;
        size_t parent = current - batch_size;

        int state[n_dimension+1] = {};
        decode(state, current);
        int currentsum = sum(state, n_dimension+1);
        int n_depletion = 0;
        int n_order = 0;

        float max_value = 0.0;
    
        struct Demand demand = demand_distribution_at_period[0];
     
        // Case 1: period < T-L-1;
        if (period < n_period- n_dimension){
               n_depletion= 0;
               n_order =0;
               if (n_capacity-1- currentsum >0){
                  n_order= n_capacity-1 - currentsum;
               }
               current_values[current] = stateValue(current, n_depletion, n_order, future_values,demand, period);  
               depletion[current] = (dp_int) n_depletion;
               order[current] = (dp_int) n_order;
        }
        // Case 2
        else {
           if (current==0 || depletion[parent]== 0){
              for (int i = 0; i <= 1; i++){        
                   int j= 0;
                   if (currentsum- i < n_capacity-1){
                      j = n_capacity-1- currentsum + i;
                   }
                   float expected_value = stateValue(current,i,j,future_values,demand, period) ;
                
                 // Simply taking the moving maximum
                   if (expected_value > max_value + 1e-8) {
                       max_value = expected_value;
                       n_depletion = i;
                       n_order = j;
                    }
               }   
               current_values[current] = max_value;
               depletion[current] = (dp_int) n_depletion;
               order[current] = (dp_int) n_order;
           }
          else{
              current_values[current] = stateValue(current,depletion[parent]+1, order[parent], future_values,demand, period);  
              depletion[current]= depletion[parent]+1;
              order[current]= order[parent];
          }
      }  
   }
}

// Plain C function to interact with kernel
// The structure is essentially the same as init_states.
// If you feel confused, start from there!
void
iter_states(float *current_values,
            dp_int *depletion,
            dp_int *order,
            float *future_values,
            int period) {

    size_t num_states = std::pow(n_capacity, n_dimension);

    // The very first state 0,0,...,0
    iter_kernel<<<1, 1>>>(current_values,
                          depletion,
                          order,
                          future_values,
                          period,
                          0, 1);

    for (size_t d = 0; d < n_dimension; d++) {

        size_t batch_size = pow(n_capacity, d);

        dim3 block_dim, grid_dim;
        get_grid_dim(&block_dim, &grid_dim, batch_size);

        for (size_t batch_idx = 1; batch_idx < n_capacity; batch_idx++) {
            iter_kernel<<<grid_dim, block_dim>>>(current_values,
                                                 depletion,
                                                 order,
                                                 future_values,
                                                 period,
                                                 batch_idx,
                                                 batch_size);
        }
    }

    hipDeviceSynchronize();
    hipDeviceSynchronize();

}

/*
 * ===  FUNCTION  ==============================================================
 *         Name:  ModelDPInit
 *  Description:  The initialization function for ModelDP
 *       @param:  the control sequence, the system information
 *      @return:  success or not
 * =============================================================================
 */
bool ModelDPInit(CommandQueue * cmd, SystemInfo * sysinfo){
    g_ModelFluidInit<<<sysinfo->get_value["num_cores"],\
                        sysinfo->get_value["core_size"]>>>\
                        (*(cmd->get_device_param_pointer), value_table);
    return true;
}       /* -----  end of function ModelDPInit  ----- */

/* :REMARKS:Sat Aug  8 12:16:39 2015:huangzonghao:
 *  shouldn't the number of index be stored in the CommandQueue????
 */
/*
 * ===  FUNCTION  ==============================================================
 *         Name:  ModelDP
 *  Description:  to update the table for one period with the dynamic programming
 *                  algorithm
 *       @param:  control sequence, system information, the index of the period
 *      @return:  success or not
 * =============================================================================
 */
bool ModelDP(CommandQueue * cmd, SystemInfo * sysinfo, int idx){

    size_t num_states = std::pow(n_capacity, n_dimension);

    float *h_current_values;
    float *h_future_values;
    dp_int *h_depletion;
    dp_int *h_order;

    checkCudaErrors(hipHostAlloc((void **)&h_current_values,
                                  sizeof(float) * num_states,
                                  hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc((void **)&h_future_values,
                                  sizeof(float) * num_states,
                                  hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc((void **)&h_depletion,
                                  sizeof(dp_int) * num_states,
                                  hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc((void **)&h_order,
                                  sizeof(dp_int) * num_states,
                                  hipHostMallocMapped));

    float *d_current_values;
    float *d_future_values;
    dp_int *d_depletion;
    dp_int *d_order;

    hipSetDeviceFlags(hipDeviceMapHost);



    checkCudaErrors(hipHostGetDevicePointer((void **)&d_current_values,
                                             (void *)h_current_values, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_future_values,
                                             (void *)h_future_values, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_depletion,
                                             (void *)h_depletion, 0));
    checkCudaErrors(hipHostGetDevicePointer((void **)&d_order,
                                             (void *)h_order, 0));

 /*   FILE *fp;
    fp = fopen("/ghome/hzhangaq/DP-parellel-computing/CCode/dp0701.log","r");
    for (int i=0; i <  num_states; i++){
        fscanf(fp,"%f", &h_future_values[i]);
    } */
    init_states(d_future_values);

  //  std::cout << "depletion,order,value" << std::endl;

    for (int period = 0; period < n_period; period++) {

        iter_states(d_current_values,
                    d_depletion,
                    d_order,
                    d_future_values,
                    period);

        // Print the results
                float *tmp = d_future_values;
        d_future_values = d_current_values;
        d_current_values = tmp;
    }
   //int state[n_dimension+1] = {};
   for (int idx = 0; idx < num_states; idx++) {
      int idxsum= 0;
      int idx_1 = idx;
      for (int i= n_dimension-1; i>= 0; i--){
          idxsum += idx_1 % n_capacity;
          idx_1 /= n_capacity;
      }
      if (idxsum <= cvalue){
         /*   int exp = std::pow(n_capacity, n_dimension-1);
            int i = idx;
            for (int k = 0; k < n_dimension; k++) {
                if (k > 0) {
                    std::cout << ',';
                }
                std::cout << i / exp;
                i %= exp;
                exp /= n_capacity;
            }
            std::cout << '\t';
            std::cout << static_cast<int>(d_depletion[idx]) << ',';
            std::cout << static_cast<int>(d_order[idx]) << ',';  */
            std::cout << std::fixed << std::setprecision(4) << d_future_values[idx];
            std::cout << '\n';
      }
    }
        std::cout << std::endl;



    checkCudaErrors(hipHostFree((void *)h_current_values));
    checkCudaErrors(hipHostFree((void *)h_future_values));
    checkCudaErrors(hipHostFree((void *)h_depletion));
    checkCudaErrors(hipHostFree((void *)h_order));

    return 0;
}       /* -----  end of function ModelDP  ----- */


/* =============================================================================
 *                         end of file model_dp.cu
 * =============================================================================
 */
